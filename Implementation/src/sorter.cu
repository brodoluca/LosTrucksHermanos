#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <iostream>
#include <deque>
#include <sys/types.h>
#include <sys/socket.h>
#include <netinet/in.h>
#include <arpa/inet.h>
#include <unistd.h>
#include <cstring>
#include "./utils/types.hpp"
#include <thread>
#include "utils/types.hpp"
#include "utils/sorter.hpp"
#include <thrust/sort.h>
#include <thrust/copy.h>
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include <stdio.h>
__global__ void enumerationSort(int *a, int *b, int N) {
  int tuid = blockIdx.x * N + threadIdx.x;
  int val = a[tuid];
  int cnt = 0;
  __shared__ int cache[100];
  
  cache[tuid] = val;
  __syncthreads();
  for ( int j = 0; j < N; ++j ) if ( val > cache[j] ) cnt++;
  b[cnt] = val;

}


void gpuSort(std::deque<TruckSocket::Message> &h_deque) {
  std::cout << "got not raw: "<<h_deque.size()<<" in dequq" << std::endl;
  if(h_deque.size()<=1)return;
  int* h_distances = (int*) malloc(h_deque.size()*sizeof(int));
  int* d_distances;hipMalloc((void**)&d_distances, h_deque.size()*sizeof(int));
  int* d_distances_sorted;hipMalloc((void**)&d_distances_sorted, h_deque.size()*sizeof(int));

  for (int i = 0; i < h_deque.size(); i++){
    h_distances[i] = h_deque.at(i)._SenderPosition;
  }
  //copy h_distances to d_distances
  hipMemcpy(d_distances, h_distances, h_deque.size()*sizeof(int), hipMemcpyHostToDevice);
  //std::cout << "copied to device" << std::endl;

  //sort d_distances
  //thrust::sort(d_distances, d_distances+h_deque.size());
  enumerationSort<<<1, h_deque.size()>>>(d_distances, d_distances_sorted, h_deque.size());
 // std::cout << "sorted" << std::endl;

  //copy d_distances to h_distances
  hipMemcpy(h_distances, d_distances_sorted, h_deque.size()*sizeof(int), hipMemcpyDeviceToHost);
  //std::cout << "copied to host" << std::endl;

  for (int i = 0; i < h_deque.size(); i++){
    std::cout << h_distances[i] << " ";
  }
  std::cout << std::endl;

  //manually sort the deque based on the sorted h_distances
  std::deque<TruckSocket::Message> temp_deque;
  for (int i = 0; i < h_deque.size(); i++){
    for (int j = 0; j < h_deque.size(); j++){
      if(h_distances[i] == h_deque.at(j)._SenderPosition){
        temp_deque.push_back(h_deque.at(j));
        break;
      }
    }
  }
  //std::cout << "manually assigned" << std::endl;


  h_deque = temp_deque;
  free(h_distances);
  hipFree(d_distances);
  hipFree(d_distances_sorted);
  return;
}

void gpuSort(std::deque<TruckSocket::RawMessage> &h_deque) {
  std::cout << "got raw: "<<h_deque.size()<<" in dequq" << std::endl;
  //returns nothing since no distance to be sorted
}

