#include <hip/hip_runtime.h>
#include <iostream>
#include <deque>
#include <sys/types.h>
#include <sys/socket.h>
#include <netinet/in.h>
#include <arpa/inet.h>
#include <unistd.h>
#include <cstring>
#include "./utils/types.hpp"
#include <thread>
#include "utils/types.hpp"
#include "utils/sorter.hpp"
#include <thrust/sort.h>
#include <thrust/copy.h>
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include <stdio.h>

void gpuSort(std::deque<TruckSocket::Message> &h_deque) {
  std::cout << "got not raw: "<<h_deque.size()<<" in dequq" << std::endl;
  if(h_deque.size()==0)return;
  int* h_distances = (int*) malloc(h_deque.size()*sizeof(int));
  int* d_distances;hipMalloc((void**)d_distances, h_deque.size()*sizeof(int));

  for (int i = 0; i < h_deque.size(); i++){
    h_distances[i] = h_deque.at(i)._SenderPosition;
  }
  //copy h_distances to d_distances
  hipMemcpy(d_distances, h_distances, h_deque.size()*sizeof(int), hipMemcpyHostToDevice);
  //sort d_distances
  thrust::sort(d_distances, d_distances+h_deque.size());

  //copy d_distances to h_distances
  hipMemcpy(h_distances, d_distances, h_deque.size()*sizeof(int), hipMemcpyDeviceToHost);
  
  //manually sort the deque based on the sorted h_distances
  std::deque<TruckSocket::Message> temp_deque;
  for (int i = 0; i < h_deque.size(); i++){
    for (int j = 0; j < h_deque.size(); j++){
      if(h_distances[i] == h_deque.at(j)._SenderPosition){
        temp_deque.push_back(h_deque.at(j));
        break;
      }
    }
  }
  h_deque = temp_deque;
  free(h_distances);
  hipFree(d_distances);

}

void gpuSort(std::deque<TruckSocket::RawMessage> &h_deque) {
  std::cout << "got raw: "<<h_deque.size()<<" in dequq" << std::endl;
  //returns nothing since no distance to be sorted
}

